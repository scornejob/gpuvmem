#include "hip/hip_runtime.h"
#include "synthesizer.cuh"
#include "imageProcessor.cuh"


long M, N, numVisibilities;
int iter=0;

float *device_Image, *device_dphi, *device_dchi2_total, *device_dS, *device_S, beam_noise, beam_bmaj, *device_noise_image, *device_weight_image;
float beam_bmin, b_noise_aux, noise_cut, MINPIX, minpix, lambda, ftol, random_probability = 1.0;
float noise_jypix, fg_scale, final_chi2, final_S, eta, robust_param;
float *host_I, sum_weights, *initial_values, *penalizators;
Telescope *telescope;

dim3 threadsPerBlockNN;
dim3 numBlocksNN;

int threadsVectorReduceNN, blocksVectorReduceNN, nopositivity = 0, verbose_flag = 0, clip_flag = 0, apply_noise = 0, print_images = 0, gridding, it_maximum, status_mod_in;
int firstgpu, selected, reg_term, total_visibilities, image_count, nPenalizators, print_errors, nMeasurementSets=0, max_number_vis;
char *output, *mempath, *out_image, *msinput, *msoutput, *inputdat, *modinput;
char *t_telescope;
float nu_0, threshold;
extern int num_gpus;

double ra, dec, crpix1, crpix2, DELTAX, DELTAY, deltau, deltav;

fitsfile *mod_in;

MSDataset *datasets;

varsPerGPU *vars_gpu;

Vars variables;

clock_t t;
double start, end;

float noise_min = 1E32;

inline bool IsGPUCapableP2P(hipDeviceProp_t *pProp)
{
  #ifdef _WIN32
        return (bool)(pProp->tccDriver ? true : false);
  #else
        return (bool)(pProp->major >= 2);
  #endif
}

std::vector<std::string> MFS::countAndSeparateStrings(char *input)
{
        char *pt;
        std::vector<std::string> ret;

        int counter = 0;
        pt = strtok(input, ",");
        while(pt!=NULL) {
                std::string s(pt);
                ret.push_back(s);
                pt = strtok (NULL, ",");
        }

        free(pt);
        return ret;
}

void MFS::configure(int argc, char **argv)
{
        if(iohandler == NULL)
        {
                iohandler = Singleton<IoFactory>::Instance().CreateIo(0);
        }

        variables = getOptions(argc, argv);
        msinput = variables.input;
        msoutput = variables.output;
        inputdat = variables.inputdat;
        modinput = variables.modin;
        out_image = variables.output_image;
        selected = variables.select;
        mempath = variables.path;
        it_maximum = variables.it_max;
        total_visibilities = 0;
        b_noise_aux = variables.noise;
        noise_cut = variables.noise_cut;
        random_probability = variables.randoms;
        eta = variables.eta;
        gridding = variables.gridding;
        nu_0 = variables.nu_0;
        robust_param = variables.robust_param;
        threshold = variables.threshold * 5.0;
        std::vector<std::string> string_values;
        std::vector<std::string> s_output_values;
        int n_outputs;


        if(strcmp(msinput, "NULL")!=0) {
                string_values = countAndSeparateStrings(msinput);
                nMeasurementSets = string_values.size();
        }else{
                printf("Datasets files were not provided\n");
                print_help();
                exit(-1);
        }

        if(strcmp(msoutput, "NULL")!=0) {
                s_output_values = countAndSeparateStrings(msoutput);
                n_outputs = s_output_values.size();
        }else{
                printf("Output/s was/were not provided\n");
                print_help();
                exit(-1);
        }

        if(n_outputs != nMeasurementSets) {
                printf("Number of input datasets should be equal to the number of output datasets\n");
                exit(-1);
        }

        datasets = (MSDataset*)malloc(nMeasurementSets*sizeof(MSDataset));

        if(verbose_flag)
                printf("Number of input datasets %d\n", nMeasurementSets);

        for(int i=0; i< nMeasurementSets; i++) {

                datasets[i].name = (char*)malloc((string_values[i].length()+1)*sizeof(char));
                datasets[i].oname = (char*)malloc((s_output_values[i].length()+1)*sizeof(char));
                strcpy(datasets[i].name, string_values[i].c_str());
                strcpy(datasets[i].oname, s_output_values[i].c_str());
        }

        string_values.clear();
        s_output_values.clear();

        if(strcmp(variables.initial_values, "NULL")!=0) {
                string_values = countAndSeparateStrings(variables.initial_values);
                image_count = string_values.size();
        }else{
                printf("Initial values for image/s were not provided\n");
                print_help();
                exit(-1);
        }

        initial_values = (float*)malloc(sizeof(float)*image_count);

        for(int i=0; i< image_count; i++)
                initial_values[i] = atof(string_values[i].c_str());

        string_values.clear();
        nu_0 = 1.0f;
        imagesChanged = 1;

        /*if(image_count > 1 && nu_0 == -1)
           {
                print_help();
                printf("for 2 or more images, nu_0 (-F) is mandatory\n");
                exit(-1);
           }*/



        /*
         *
         * Create directory to save images for each iterations
         */
        struct stat st = {0};
        if(print_images)
                if(stat(mempath, &st) == -1) mkdir(mempath,0700);

        hipDeviceProp_t dprop[num_gpus];

        if(verbose_flag) {
                printf("Number of host CPUs:\t%d\n", omp_get_num_procs());
                printf("Number of CUDA devices:\t%d\n", num_gpus);


                for(int i = 0; i < num_gpus; i++) {
                        hipGetDeviceProperties(&dprop[i], i);

                        printf("> GPU%d = \"%15s\" %s capable of Peer-to-Peer (P2P)\n", i, dprop[i].name, (IsGPUCapableP2P(&dprop[i]) ? "IS " : "NOT"));

                        //printf("   %d: %s\n", i, dprop.name);
                }
                printf("---------------------------\n");
        }

        hipGetDeviceProperties(&dprop[0], 0);
        if(variables.blockSizeX*variables.blockSizeY > dprop[0].maxThreadsPerBlock || variables.blockSizeV > dprop[0].maxThreadsPerBlock) {
                printf("Block size X: %d\n", variables.blockSizeX);
                printf("Block size Y: %d\n", variables.blockSizeY);
                printf("Block size X*Y: %d\n", variables.blockSizeX*variables.blockSizeY);
                printf("Block size V: %d\n", variables.blockSizeV);
                printf("ERROR. The maximum threads per block cannot be greater than %d\n", dprop[0].maxThreadsPerBlock);
                exit(-1);
        }

        if(variables.blockSizeX > dprop[0].maxThreadsDim[0] || variables.blockSizeY > dprop[0].maxThreadsDim[1] || variables.blockSizeV > dprop[0].maxThreadsDim[0]) {
                printf("Block size X: %d\n", variables.blockSizeX);
                printf("Block size Y: %d\n", variables.blockSizeY);
                printf("Block size V: %d\n", variables.blockSizeV);
                printf("ERROR. The size of the blocksize cannot exceed X: %d Y: %d Z: %d\n", dprop[0].maxThreadsDim[0], dprop[0].maxThreadsDim[1], dprop[0].maxThreadsDim[2]);
                exit(-1);
        }

        if(selected > num_gpus || selected < 0) {
                printf("ERROR. THE SELECTED GPU DOESN'T EXIST\n");
                exit(-1);
        }

        readInputDat(inputdat);

        int n_telescopes;
        if(strcmp(t_telescope, "NULL")!=0) {
                string_values = countAndSeparateStrings(t_telescope);
                n_telescopes = string_values.size();
        }else{
                printf("Telescope codes were not provided\n");
                print_help();
                exit(-1);
        }

        if(n_telescopes != nMeasurementSets) {
                printf("Number of telescope codes cannot be different to the number of Measurement Sets\n");
                print_help();
                exit(-1);
        }

        int telescope;
        for(int i=0; i< nMeasurementSets; i++) {

                telescope = atoi(string_values[i].c_str());
                init_beam(telescope, &datasets[i].antenna_diameter, &datasets[i].pb_factor, &datasets[i].pb_cutoff);
        }


        canvasVariables canvas_vars = iohandler->IoreadCanvas(modinput, mod_in, b_noise_aux, status_mod_in, verbose_flag);

        M = canvas_vars.M;
        N = canvas_vars.N;
        DELTAX = canvas_vars.DELTAX;
        DELTAY = canvas_vars.DELTAY;
        ra = canvas_vars.ra;
        dec = canvas_vars.dec;
        crpix1 = canvas_vars.crpix1;
        crpix2 = canvas_vars.crpix2;
        beam_bmaj = canvas_vars.beam_bmaj;
        beam_bmin = canvas_vars.beam_bmin;
        beam_noise = canvas_vars.beam_noise;

        if(verbose_flag)
                printf("Counting data for memory allocation\n");

        for(int i=0; i<nMeasurementSets; i++) {
                datasets[i].data = iohandler->IocountVisibilities(datasets[i].name, datasets[i].fields, gridding);
        }

        if(verbose_flag) {
                for(int i=0; i<nMeasurementSets; i++) {
                        printf("Dataset %s\n", datasets[i].name);
                        printf("\tNumber of fields = %d\n", datasets[i].data.nfields);
                        printf("\tNumber of frequencies = %d\n", datasets[i].data.total_frequencies);
                        printf("\tNumber of Stokes = %d\n", datasets[i].data.nstokes);
                }
        }

        if(strcmp(variables.penalization_factors, "NULL")!=0) {

                string_values = countAndSeparateStrings(variables.penalization_factors);
                nPenalizators = string_values.size();
                penalizators = (float*)malloc(sizeof(float)*nPenalizators);
                for(int i = 0; i < nPenalizators; i++)
                        penalizators[i] = atof(string_values[i].c_str());

        }else{
                printf("No regularization factors provided\n");
        }
        string_values.clear();

        num_gpus = 1;
        firstgpu = selected;
        //printf("number of FINAL host CPUs:\t%d\n", omp_get_num_procs());
        if(verbose_flag) {
                printf("Number of CUDA devices and threads: \t%d\n", num_gpus);
        }


        vars_gpu = (varsPerGPU*)malloc(num_gpus*sizeof(varsPerGPU));

        for(int d=0; d<nMeasurementSets; d++) {
                for(int f=0; f<datasets[d].data.nfields; f++) {
                        datasets[d].fields[f].visibilities = (Vis**)malloc(datasets[d].data.total_frequencies*sizeof(Vis*));
                        datasets[d].fields[f].device_visibilities = (Vis**)malloc(datasets[d].data.total_frequencies*sizeof(Vis*));
                        datasets[d].fields[f].nu = (float*)malloc(datasets[d].data.total_frequencies*sizeof(float));
                        if(gridding) {
                                datasets[d].fields[f].gridded_visibilities = (Vis **) malloc(datasets[d].data.total_frequencies * sizeof(Vis * ));
                                datasets[d].fields[f].backup_visibilities = (Vis **) malloc(datasets[d].data.total_frequencies * sizeof(Vis * ));
                        }

                        for(int i=0; i<datasets[d].data.total_frequencies; i++) {
                                datasets[d].fields[f].visibilities[i] = (Vis*)malloc(datasets[d].data.nstokes*sizeof(Vis));
                                datasets[d].fields[f].device_visibilities[i] = (Vis*)malloc(datasets[d].data.nstokes*sizeof(Vis));

                                if(gridding) {
                                        datasets[d].fields[f].gridded_visibilities[i] = (Vis*)malloc(datasets[d].data.nstokes*sizeof(Vis));
                                        datasets[d].fields[f].backup_visibilities[i] = (Vis*)malloc(datasets[d].data.nstokes*sizeof(Vis));
                                }
                        }
                }
        }


        //ALLOCATE MEMORY AND GET TOTAL NUMBER OF VISIBILITIES
        for(int d=0; d<nMeasurementSets; d++) {
                for(int f=0; f<datasets[d].data.nfields; f++) {
                        for(int i=0; i < datasets[d].data.total_frequencies; i++) {
                                for(int s=0; s<datasets[d].data.nstokes; s++) {
                                        datasets[d].fields[f].visibilities[i][s].uvw = (double3*)malloc(datasets[d].fields[f].numVisibilitiesPerFreqPerStoke[i][s]*sizeof(double3));
                                        datasets[d].fields[f].visibilities[i][s].weight = (float*)malloc(datasets[d].fields[f].numVisibilitiesPerFreqPerStoke[i][s]*sizeof(float));
                                        datasets[d].fields[f].visibilities[i][s].Vo = (hipfftComplex*)malloc(datasets[d].fields[f].numVisibilitiesPerFreqPerStoke[i][s]*sizeof(hipfftComplex));
                                        datasets[d].fields[f].visibilities[i][s].Vm = (hipfftComplex*)malloc(datasets[d].fields[f].numVisibilitiesPerFreqPerStoke[i][s]*sizeof(hipfftComplex));

                                        if(gridding)
                                        {
                                                datasets[d].fields[f].gridded_visibilities[i][s].uvw = (double3*)malloc(M*N*sizeof(double3));
                                                datasets[d].fields[f].gridded_visibilities[i][s].weight = (float*)malloc(M*N*sizeof(float));
                                                datasets[d].fields[f].gridded_visibilities[i][s].S = (int*)malloc(M*N*sizeof(int));
                                                datasets[d].fields[f].gridded_visibilities[i][s].Vo = (hipfftComplex*)malloc(M*N*sizeof(hipfftComplex));
                                                datasets[d].fields[f].gridded_visibilities[i][s].Vm = (hipfftComplex*)malloc(M*N*sizeof(hipfftComplex));

                                                memset(datasets[d].fields[f].gridded_visibilities[i][s].uvw, 0, M*N*sizeof(double3));
                                                memset(datasets[d].fields[f].gridded_visibilities[i][s].weight, 0, M*N*sizeof(float));
                                                memset(datasets[d].fields[f].gridded_visibilities[i][s].S, 0, M*N*sizeof(int));
                                                memset(datasets[d].fields[f].gridded_visibilities[i][s].Vo, 0, M*N*sizeof(hipfftComplex));
                                                memset(datasets[d].fields[f].gridded_visibilities[i][s].Vm, 0, M*N*sizeof(hipfftComplex));

                                                //Allocate memory to backup original visibilities after gridding
                                                datasets[d].fields[f].backup_visibilities[i][s].uvw = (double3*)malloc(datasets[d].fields[f].numVisibilitiesPerFreqPerStoke[i][s]*sizeof(double3));
                                                datasets[d].fields[f].backup_visibilities[i][s].weight = (float*)malloc(datasets[d].fields[f].numVisibilitiesPerFreqPerStoke[i][s]*sizeof(float));
                                                datasets[d].fields[f].backup_visibilities[i][s].Vo = (hipfftComplex*)malloc(datasets[d].fields[f].numVisibilitiesPerFreqPerStoke[i][s]*sizeof(hipfftComplex));
                                                datasets[d].fields[f].backup_visibilities[i][s].Vm = (hipfftComplex*)malloc(datasets[d].fields[f].numVisibilitiesPerFreqPerStoke[i][s]*sizeof(hipfftComplex));

                                                memset(datasets[d].fields[f].backup_visibilities[i][s].uvw, 0, datasets[d].fields[f].numVisibilitiesPerFreqPerStoke[i][s]*sizeof(double3));
                                                memset(datasets[d].fields[f].backup_visibilities[i][s].weight, 0, datasets[d].fields[f].numVisibilitiesPerFreqPerStoke[i][s]*sizeof(float));
                                                memset(datasets[d].fields[f].backup_visibilities[i][s].Vo, 0, datasets[d].fields[f].numVisibilitiesPerFreqPerStoke[i][s]*sizeof(hipfftComplex));
                                                memset(datasets[d].fields[f].backup_visibilities[i][s].Vm, 0, datasets[d].fields[f].numVisibilitiesPerFreqPerStoke[i][s]*sizeof(hipfftComplex));

                                        }
                                }
                        }
                }
        }

        if(verbose_flag) {
                printf("Reading visibilities and FITS input files...\n");
        }

        for(int d=0; d<nMeasurementSets; d++) {
                if(apply_noise) {
                        iohandler->IoreadMS(datasets[d].name, datasets[d].fields, datasets[d].data, true, false, random_probability);
                }else{
                        iohandler->IoreadMS(datasets[d].name, datasets[d].fields, datasets[d].data, false, false, random_probability);
                }
        }

        this->visibilities = new Visibilities();
        this->visibilities->setMSDataset(datasets);
        this->visibilities->setNDatasets(nMeasurementSets);

        double deltax = RPDEG_D*DELTAX; //radians
        double deltay = RPDEG_D*DELTAY; //radians
        deltau = 1.0 / (M * deltax);
        deltav = 1.0 / (N * deltay);

        if(gridding) {
                printf("Doing gridding\n");
                omp_set_num_threads(gridding);
                for(int d=0; d<nMeasurementSets; d++)
                        do_gridding(datasets[d].fields, &datasets[d].data, deltau, deltav, M, N, robust_param);

                omp_set_num_threads(num_gpus);
        }
}

void MFS::setDevice()
{
        double deltax = RPDEG_D*DELTAX; //radians
        double deltay = RPDEG_D*DELTAY; //radians
        deltau = 1.0 / (M * deltax);
        deltav = 1.0 / (N * deltay);

        if(verbose_flag) {
                printf("MS File Successfully Read\n");
                if(beam_noise == -1) {
                        printf("Beam noise wasn't provided by the user... Calculating...\n");
                }
        }

        for(int d=0; d<nMeasurementSets; d++) {
                sum_weights = calculateNoise(datasets[d].fields, datasets[d].data, &total_visibilities, variables.blockSizeV, gridding);
        }

        this->visibilities->setTotalVisibilities(total_visibilities);

        for(int d=0; d<nMeasurementSets; d++) {
                for(int f=0; f<datasets[d].data.nfields; f++) {
                        hipSetDevice(selected);
                        for(int i=0; i<datasets[d].data.total_frequencies; i++) {
                                for(int s=0; s<datasets[d].data.nstokes; s++) {
                                        gpuErrchk(hipMalloc(&datasets[d].fields[f].device_visibilities[i][s].uvw,
                                                             sizeof(double3) * datasets[d].fields[f].numVisibilitiesPerFreqPerStoke[i][s]));
                                        gpuErrchk(hipMalloc(&datasets[d].fields[f].device_visibilities[i][s].Vo,
                                                             sizeof(hipfftComplex) * datasets[d].fields[f].numVisibilitiesPerFreqPerStoke[i][s]));
                                        gpuErrchk(hipMalloc(&datasets[d].fields[f].device_visibilities[i][s].weight,
                                                             sizeof(float) * datasets[d].fields[f].numVisibilitiesPerFreqPerStoke[i][s]));
                                        gpuErrchk(hipMalloc(&datasets[d].fields[f].device_visibilities[i][s].Vm,
                                                             sizeof(hipfftComplex) * datasets[d].fields[f].numVisibilitiesPerFreqPerStoke[i][s]));
                                        gpuErrchk(hipMalloc(&datasets[d].fields[f].device_visibilities[i][s].Vr,
                                                             sizeof(hipfftComplex) * datasets[d].fields[f].numVisibilitiesPerFreqPerStoke[i][s]));
                                        gpuErrchk(hipMemcpy(datasets[d].fields[f].device_visibilities[i][s].uvw, datasets[d].fields[f].visibilities[i][s].uvw,
                                                             sizeof(double3) * datasets[d].fields[f].numVisibilitiesPerFreqPerStoke[i][s],
                                                             hipMemcpyHostToDevice));

                                        gpuErrchk(hipMemcpy(datasets[d].fields[f].device_visibilities[i][s].weight,
                                                             datasets[d].fields[f].visibilities[i][s].weight,
                                                             sizeof(float) * datasets[d].fields[f].numVisibilitiesPerFreqPerStoke[i][s],
                                                             hipMemcpyHostToDevice));

                                        gpuErrchk(hipMemcpy(datasets[d].fields[f].device_visibilities[i][s].Vo, datasets[d].fields[f].visibilities[i][s].Vo,
                                                             sizeof(hipfftComplex) * datasets[d].fields[f].numVisibilitiesPerFreqPerStoke[i][s],
                                                             hipMemcpyHostToDevice));

                                        gpuErrchk(hipMemset(datasets[d].fields[f].device_visibilities[i][s].Vr, 0,
                                                             sizeof(hipfftComplex) * datasets[d].fields[f].numVisibilitiesPerFreqPerStoke[i][s]));
                                        gpuErrchk(hipMemset(datasets[d].fields[f].device_visibilities[i][s].Vm, 0,
                                                             sizeof(hipfftComplex) * datasets[d].fields[f].numVisibilitiesPerFreqPerStoke[i][s]));
                                }
                        }

                        gpuErrchk(hipMalloc((void**)&datasets[d].fields[f].atten_image, sizeof(float)*M*N));
                        gpuErrchk(hipMemset(datasets[d].fields[f].atten_image, 0, sizeof(float)*M*N));

                }
        }


        max_number_vis = 0;
        for(int d=0; d<nMeasurementSets; d++) {
                if(datasets[d].data.max_number_visibilities_in_channel_and_stokes > max_number_vis)
                        max_number_vis = datasets[d].data.max_number_visibilities_in_channel_and_stokes;
        }

        if(max_number_vis == 0) {
                printf("Max number of visibilities cannot be zero for image synthesis\n");
                exit(-1);
        }

        this->visibilities->setMaxNumberVis(max_number_vis);

        for(int g=0; g<num_gpus; g++) {
                hipSetDevice((g%num_gpus) + firstgpu);
                gpuErrchk(hipMalloc((void**)&vars_gpu[g].device_dchi2, sizeof(float)*M*N));
                gpuErrchk(hipMemset(vars_gpu[g].device_dchi2, 0, sizeof(float)*M*N));

                gpuErrchk(hipMalloc(&vars_gpu[g].device_chi2, sizeof(float)*max_number_vis));
                gpuErrchk(hipMemset(vars_gpu[g].device_chi2, 0, sizeof(float)*max_number_vis));
        }

        //Declaring block size and number of blocks for Image
        dim3 threads(variables.blockSizeX, variables.blockSizeY);
        dim3 blocks(M/threads.x, N/threads.y);
        threadsPerBlockNN = threads;
        numBlocksNN = blocks;

        noise_jypix = beam_noise / (PI * beam_bmaj * beam_bmin / (4 * log(2) ));

        /////////////////////////////////////////////////////CALCULATE DIRECTION COSINES/////////////////////////////////////////////////
        double raimage = ra * RPDEG_D;
        double decimage = dec * RPDEG_D;

        if(verbose_flag) {
                printf("FITS: Ra: %.16e (rad), dec: %.16e (rad)\n", raimage, decimage);
                printf("FITS: Center pix: (%lf,%lf)\n", crpix1-1, crpix2-1);
        }

        double lobs, mobs, lphs, mphs;
        double dcosines_l_pix_ref, dcosines_m_pix_ref, dcosines_l_pix_phs, dcosines_m_pix_phs;
        for(int d=0; d<nMeasurementSets; d++) {
                if(verbose_flag)
                        printf("Dataset: %s\n", datasets[d].name);
                for(int f=0; f<datasets[d].data.nfields; f++) {

                        direccos(datasets[d].fields[f].ref_ra, datasets[d].fields[f].ref_dec, raimage, decimage, &lobs,  &mobs);
                        direccos(datasets[d].fields[f].phs_ra, datasets[d].fields[f].phs_dec, raimage, decimage, &lphs,  &mphs);

                        dcosines_l_pix_ref = lobs/ -deltax; // Radians to pixels
                        dcosines_m_pix_ref = mobs/fabs(deltay); // Radians to pixels

                        dcosines_l_pix_phs = lphs/ -deltax; // Radians to pixels
                        dcosines_m_pix_phs = mphs/fabs(deltay); // Radians to pixels

                        if(verbose_flag)
                        {
                                printf("Ref: l (pix): %e, m (pix): %e\n", dcosines_l_pix_ref, dcosines_m_pix_ref);
                                printf("Phase: l (pix): %e, m (pix): %e\n", dcosines_l_pix_phs, dcosines_m_pix_phs);

                        }


                        datasets[d].fields[f].ref_xobs = (crpix1 - 1.0f) + dcosines_l_pix_ref;// + 6.0f;
                        datasets[d].fields[f].ref_yobs = (crpix2 - 1.0f) + dcosines_m_pix_ref;// - 7.0f;

                        datasets[d].fields[f].phs_xobs = (crpix1 - 1.0f) + dcosines_l_pix_phs;// + 5.0f;
                        datasets[d].fields[f].phs_yobs = (crpix2 - 1.0f) + dcosines_m_pix_phs;// - 7.0f;


                        if(verbose_flag) {
                                printf("Ref: Field %d - Ra: %.16e (rad), dec: %.16e (rad), x0: %f (pix), y0: %f (pix)\n", f, datasets[d].fields[f].ref_ra, datasets[d].fields[f].ref_dec,
                                       datasets[d].fields[f].ref_xobs, datasets[d].fields[f].ref_yobs);
                                printf("Phase: Field %d - Ra: %.16e (rad), dec: %.16e (rad), x0: %f (pix), y0: %f (pix)\n", f, datasets[d].fields[f].phs_ra, datasets[d].fields[f].phs_dec,
                                       datasets[d].fields[f].phs_xobs, datasets[d].fields[f].phs_yobs);
                        }

                        if(datasets[d].fields[f].ref_xobs < 0 || datasets[d].fields[f].ref_xobs >= M || datasets[d].fields[f].ref_xobs < 0 || datasets[d].fields[f].ref_yobs >= N) {
                                printf("Dataset: %s\n", datasets[d].name);
                                printf("Pointing reference center (%f,%f) is outside the range of the image\n", datasets[d].fields[f].ref_xobs, datasets[d].fields[f].ref_yobs);
                                goToError();
                        }

                        if(datasets[d].fields[f].phs_xobs < 0 || datasets[d].fields[f].phs_xobs >= M || datasets[d].fields[f].phs_xobs < 0 || datasets[d].fields[f].phs_yobs >= N) {
                                printf("Dataset: %s\n", datasets[d].name);
                                printf("Pointing phase center (%f,%f) is outside the range of the image\n", datasets[d].fields[f].phs_xobs, datasets[d].fields[f].phs_yobs);
                                goToError();
                        }
                }
        }
        ////////////////////////////////////////////////////////MAKE STARTING IMAGE////////////////////////////////////////////////////////

        host_I = (float*)malloc(M*N*sizeof(float)*image_count);

        for(int i=0; i<M; i++) {
                for(int j=0; j<N; j++) {
                        for(int k=0; k<image_count; k++) {
                                host_I[N*M*k+N*i+j] = initial_values[k];
                        }
                }
        }

        ////////////////////////////////////////////////CUDA MEMORY ALLOCATION FOR DEVICE///////////////////////////////////////////////////

        for(int g=0; g<num_gpus; g++) {
                hipSetDevice((g%num_gpus) + firstgpu);
                gpuErrchk(hipMalloc((void**)&vars_gpu[g].device_V, sizeof(hipfftComplex)*M*N*image_count));
                gpuErrchk(hipMalloc((void**)&vars_gpu[g].device_IQUV_nu, sizeof(hipfftComplex)*M*N*image_count));
        }


        hipSetDevice(firstgpu);

        gpuErrchk(hipMalloc((void**)&device_Image, sizeof(float)*M*N*image_count));
        gpuErrchk(hipMemset(device_Image, 0, sizeof(float)*M*N*image_count));

        gpuErrchk(hipMemcpy(device_Image, host_I, sizeof(float)*N*M*image_count, hipMemcpyHostToDevice));

        gpuErrchk(hipMalloc((void**)&device_noise_image, sizeof(float)*M*N));
        gpuErrchk(hipMemset(device_noise_image, 0, sizeof(float)*M*N));

        gpuErrchk(hipMalloc((void**)&device_weight_image, sizeof(float)*M*N));
        gpuErrchk(hipMemset(device_weight_image, 0, sizeof(float)*M*N));



        for(int g=0; g<num_gpus; g++) {
                hipSetDevice((g%num_gpus) + firstgpu);
                gpuErrchk(hipMemset(vars_gpu[g].device_V, 0, sizeof(hipfftComplex)*M*N*image_count));
                gpuErrchk(hipMemset(vars_gpu[g].device_I_nu, 0, sizeof(hipfftComplex)*M*N*image_count));

        }

        /////////// MAKING IMAGE OBJECT /////////////
        image = new Image(device_Image, image_count);
        imageMap *functionPtr = (imageMap*)malloc(sizeof(imageMap)*image_count);
        image->setFunctionMapping(functionPtr);

        for(int i = 0; i < image_count; i++)
        {
                if(nopositivity)
                {
                        functionPtr[i].evaluateXt = defaultEvaluateXt;
                        functionPtr[i].newP = defaultNewP;
                }else{
                        if(!i)
                        {
                                functionPtr[i].evaluateXt = particularEvaluateXt;
                                functionPtr[i].newP = particularNewP;
                        }else{
                                functionPtr[i].evaluateXt = defaultEvaluateXt;
                                functionPtr[i].newP = defaultNewP;
                        }
                }
        }


        initFFT(vars_gpu, M, N, firstgpu, num_gpus);

        //Time is taken from first kernel
        t = clock();
        start = omp_get_wtime();
        for(int d=0; d<nMeasurementSets; d++) {
                for(int f=0; f < datasets[d].data.nfields; f++) {
                        hipSetDevice(selected);
                        for(int i=0; i<datasets[d].data.total_frequencies; i++) {
                                for(int s=0; s<datasets[d].data.nstokes; s++) {
                                        hermitianSymmetry << < datasets[d].fields[f].visibilities[i][s].numBlocksUV,
                                                datasets[d].fields[f].visibilities[i][s].threadsPerBlockUV >> >
                                        (datasets[d].fields[f].device_visibilities[i][s].uvw, datasets[d].fields[f].device_visibilities[i][s].Vo, datasets[d].fields[f].nu[i], datasets[d].fields[f].numVisibilitiesPerFreqPerStoke[i][s]);
                                        gpuErrchk(hipDeviceSynchronize());
                                }
                        }
                }

                hipSetDevice(selected);
                for(int f=0; f<datasets[d].data.nfields; f++) {
                        for(int i=0; i<datasets[d].data.total_frequencies; i++) {
                                if(datasets[d].fields[f].numVisibilitiesPerFreq[i] > 0) {
                                        total_attenuation<<<numBlocksNN, threadsPerBlockNN>>>(datasets[d].fields[f].atten_image, datasets[d].antenna_diameter, datasets[d].pb_factor, datasets[d].pb_cutoff, datasets[d].fields[f].nu[i], datasets[d].fields[f].ref_xobs, datasets[d].fields[f].ref_yobs, DELTAX, DELTAY, N);
                                        gpuErrchk(hipDeviceSynchronize());
                                }
                        }
                }


                for(int f=0; f<datasets[d].data.nfields; f++) {
                        if(datasets[d].fields[f].valid_frequencies > 0) {
                                if(num_gpus == 1) {
                                        hipSetDevice(selected);
                                        mean_attenuation<<<numBlocksNN, threadsPerBlockNN>>>(datasets[d].fields[f].atten_image, datasets[d].fields[f].valid_frequencies, N);
                                        gpuErrchk(hipDeviceSynchronize());
                                }else{
                                        hipSetDevice(firstgpu);
                                        mean_attenuation<<<numBlocksNN, threadsPerBlockNN>>>(datasets[d].fields[f].atten_image, datasets[d].fields[f].valid_frequencies, N);
                                        gpuErrchk(hipDeviceSynchronize());
                                }
                                if(print_images) {
                                        std::string atten_name =  "dataset_" + std::to_string(d) + "_atten";
                                        iohandler->IoPrintImageIteration(datasets[d].fields[f].atten_image, mod_in, mempath, atten_name.c_str(), "", f, 0, 1.0, M, N);
                                }
                        }
                }
        }



        if(num_gpus == 1) {
                hipSetDevice(selected);
        }else{
                hipSetDevice(firstgpu);
        }

        for(int d=0; d<nMeasurementSets; d++) {
                for(int f=0; f<datasets[d].data.nfields; f++) {
                        weight_image<<<numBlocksNN, threadsPerBlockNN>>>(device_weight_image, datasets[d].fields[f].atten_image, noise_jypix, N);
                        gpuErrchk(hipDeviceSynchronize());
                }
        }

        noise_image<<<numBlocksNN, threadsPerBlockNN>>>(device_noise_image, device_weight_image, noise_jypix, N);
        gpuErrchk(hipDeviceSynchronize());
        if(print_images)
                iohandler->IoPrintImage(device_noise_image, mod_in, mempath, "noise.fits", "", 0, 0, 1.0, M, N);


        float *host_noise_image = (float*)malloc(M*N*sizeof(float));
        gpuErrchk(hipMemcpy2D(host_noise_image, sizeof(float), device_noise_image, sizeof(float), sizeof(float), M*N, hipMemcpyDeviceToHost));
        float noise_min = *std::min_element(host_noise_image,host_noise_image+(M*N));

        fg_scale = noise_min;
        noise_cut = noise_cut * noise_min;
        if(verbose_flag) {
                printf("fg_scale = %e\n", fg_scale);
                printf("noise (Jy/pix) = %e\n", noise_jypix);
        }
        free(host_noise_image);
        hipFree(device_weight_image);
        for(int d=0; d<nMeasurementSets; d++) {
                for(int f=0; f<datasets[d].data.nfields; f++) {
                        hipFree(datasets[d].fields[f].atten_image);
                }
        }
};

void MFS::run()
{
        //printf("\n\nStarting Fletcher Reeves Polak Ribiere method (Conj. Grad.)\n\n");
        printf("\n\nStarting Optimizator\n");
        optimizator->getObjectiveFuntion()->setIo(iohandler);
        optimizator->getObjectiveFuntion()->setPrintImages(print_images);
        //optimizator->getObjectiveFuntion()->setIoOrderIterations(IoOrderIterations);
        if(this->Order == NULL) {
                if(imagesChanged)
                {
                        optimizator->setImage(image);
                        optimizator->minimizate();
                }
        }else{
                (this->Order)(optimizator, image);
        }

        t = clock() - t;
        end = omp_get_wtime();
        printf("Minimization ended successfully\n\n");
        printf("Iterations: %d\n", iter);
        printf("chi2: %f\n", final_chi2);
        printf("0.5*chi2: %f\n", 0.5*final_chi2);
        printf("Total visibilities: %d\n", total_visibilities);
        printf("Reduced-chi2 (Num visibilities): %f\n", (0.5*final_chi2)/total_visibilities);
        printf("Reduced-chi2 (Weights sum): %f\n", (0.5*final_chi2)/sum_weights);
        printf("S: %f\n", final_S);
        if(reg_term != 1) {
                printf("Normalized S: %f\n", final_S/(M*N));
        }else{
                printf("Normalized S: %f\n", final_S/(M*M*N*N));
        }
        printf("lambda*S: %f\n\n", lambda*final_S);
        double time_taken = ((double)t)/CLOCKS_PER_SEC;
        double wall_time = end-start;
        printf("Total CPU time: %lf\n", time_taken);
        printf("Wall time: %lf\n\n\n", wall_time);

        if(strcmp(variables.ofile,"NULL") != 0) {
                FILE *outfile = fopen(variables.ofile, "w");
                if (outfile == NULL)
                {
                        printf("Error opening output file!\n");
                        goToError();
                }

                fprintf(outfile, "Iterations: %d\n", iter);
                fprintf(outfile, "chi2: %f\n", final_chi2);
                fprintf(outfile, "0.5*chi2: %f\n", 0.5*final_chi2);
                fprintf(outfile, "Total visibilities: %d\n", total_visibilities);
                fprintf(outfile, "Reduced-chi2 (Num visibilities): %f\n", (0.5*final_chi2)/total_visibilities);
                fprintf(outfile, "Reduced-chi2 (Weights sum): %f\n", (0.5*final_chi2)/sum_weights);
                fprintf(outfile, "S: %f\n", final_S);
                if(reg_term != 1) {
                        fprintf(outfile, "Normalized S: %f\n", final_S/(M*N));
                }else{
                        fprintf(outfile, "Normalized S: %f\n", final_S/(M*M*N*N));
                }
                fprintf(outfile, "lambda*S: %f\n", lambda*final_S);
                fprintf(outfile, "Wall time: %lf", wall_time);
                fclose(outfile);
        }
        //Pass residuals to host
        printf("Saving final image to disk\n");
        if(IoOrderEnd == NULL) {
                iohandler->IoPrintImage(image->getImage(), mod_in, "", out_image, "JY/PIXEL", iter, 0, fg_scale, M, N);
                iohandler->IoPrintImage(image->getImage(), mod_in, "", "alpha.fits", "", iter, 1, 1.0, M, N);
        }else{
                (IoOrderEnd)(image->getImage(), iohandler);
        }

        if(print_errors) /* flag for print error image */
        {
                if(this->error == NULL)
                {
                        this->error = Singleton<ErrorFactory>::Instance().CreateError(0);
                }
                /* code to calculate error */
                /* make void * params */
                printf("Calculating Error Images\n");
                this->error->calculateErrorImage(this->image, this->visibilities);
                if(IoOrderError == NULL) {
                        iohandler->IoPrintImage(image->getErrorImage(), mod_in, "", "error_Inu_0.fits", "JY/PIXEL", iter, 0, 1.0, M, N);
                        iohandler->IoPrintImage(image->getErrorImage(), mod_in, "", "error_alpha.fits", "", iter, 1, 1.0, M, N);
                }else{
                        (IoOrderError)(image->getErrorImage(), iohandler);
                }

        }

        if(!gridding)
        {
                //Saving residuals to disk
                for(int d=0; d<nMeasurementSets; d++) {
                        residualsToHost(datasets[d].fields, datasets[d].data, num_gpus, firstgpu);
                }
        }else{
                double deltax = RPDEG_D*DELTAX; //radians
                double deltay = RPDEG_D*DELTAY; //radians
                deltau = 1.0 / (M * deltax);
                deltav = 1.0 / (N * deltay);

                printf("Visibilities are gridded, we will need to de-grid to save them in a Measurement Set File\n");
                omp_set_num_threads(gridding);
                for(int d=0; d<nMeasurementSets; d++)
                        degridding(datasets[d].fields, datasets[d].data, deltau, deltav, num_gpus, firstgpu, variables.blockSizeV, M, N);

                omp_set_num_threads(num_gpus);

                for(int d=0; d<nMeasurementSets; d++)
                        residualsToHost(datasets[d].fields, datasets[d].data, num_gpus, firstgpu);

        }

        printf("Saving residuals to MS...\n");
        for(int d=0; d<nMeasurementSets; d++)
                iohandler->IowriteMS(datasets[d].name, datasets[d].oname, datasets[d].fields, datasets[d].data, random_probability, false, false, false, verbose_flag);

        printf("Residuals saved.\n");


};

void MFS::unSetDevice()
{
        //Free device and host memory
        printf("Freeing device memory\n");
        if(num_gpus == 1) {
                hipSetDevice(selected);
        }else{
                hipSetDevice(firstgpu);
        }

        for(int d=0; d<nMeasurementSets; d++) {
                for(int f=0; f<datasets[d].data.nfields; f++) {
                        for(int i=0; i<datasets[d].data.total_frequencies; i++) {

                                if(num_gpus > 1) {
                                        hipSetDevice((i%num_gpus) + firstgpu);
                                }
                                for(int s=0; s<datasets[d].data.nstokes; s++) {
                                        hipFree(datasets[d].fields[f].device_visibilities[i][s].uvw);
                                        hipFree(datasets[d].fields[f].device_visibilities[i][s].weight);
                                        hipFree(datasets[d].fields[f].device_visibilities[i][s].Vr);
                                        hipFree(datasets[d].fields[f].device_visibilities[i][s].Vm);
                                        hipFree(datasets[d].fields[f].device_visibilities[i][s].Vo);
                                }

                        }
                }
        }

        printf("Freeing cuFFT plans\n");
        for(int g=0; g<num_gpus; g++) {
                hipSetDevice((g%num_gpus) + firstgpu);
                hipfftDestroy(vars_gpu[g].plan);
        }

        printf("Freeing host memory\n");
        for(int d=0; d<nMeasurementSets; d++) {
                for(int f=0; f<datasets[d].data.nfields; f++) {
                        for(int i=0; i<datasets[d].data.total_frequencies; i++) {
                                for(int s=0; s<datasets[d].data.nstokes; s++) {
                                        if (datasets[d].fields[f].numVisibilitiesPerFreqPerStoke[i][s] > 0) {
                                                free(datasets[d].fields[f].visibilities[i][s].uvw);
                                                free(datasets[d].fields[f].visibilities[i][s].weight);
                                                free(datasets[d].fields[f].visibilities[i][s].Vo);
                                                free(datasets[d].fields[f].visibilities[i][s].Vm);
                                        }
                                }
                        }
                }
        }



        hipFree(device_Image);

        for(int g=0; g<num_gpus; g++) {
                hipSetDevice((g%num_gpus) + firstgpu);
                hipFree(vars_gpu[g].device_V);
                hipFree(vars_gpu[g].device_I_nu);
        }


        hipSetDevice(firstgpu);


        hipFree(device_noise_image);

        hipFree(device_dphi);
        hipFree(device_dchi2_total);
        hipFree(device_dS);

        hipFree(device_S);

        free(host_I);
        free(msinput);
        free(t_telescope);
        free(msoutput);
        free(modinput);

        for(int i=0; i< nMeasurementSets; i++) {
                free(datasets[i].name);
                free(datasets[i].oname);
        }

        iohandler->IocloseCanvas(mod_in);
};

namespace {
Synthesizer* CreateMFS()
{
        return new MFS;
}
const int MFSID = 0;
const bool RegisteredMFS = Singleton<SynthesizerFactory>::Instance().RegisterSynthesizer(MFSID, CreateMFS);
};
